#include "hip/hip_runtime.h"
#include "../include/cuda_basic.h"
#include "../include/class_def.h"
#include <hip/hip_math_constants.h>

__device__ bool __CUDA_KERNEL_ComputeFeasibleForAngle(
    float3 *frame,
    int *classCost,
    int x,
    int z,
    float angle_radians,
    int width,
    int height,
    int min_dist_x,
    int min_dist_z,
    int lower_bound_ego_x,
    int lower_bound_ego_z,
    int upper_bound_ego_x,
    int upper_bound_ego_z)
{
    float c = cosf(angle_radians);
    float s = sinf(angle_radians);

    for (int j = -min_dist_z; j <= min_dist_z; j++)
        for (int i = -min_dist_x; i <= min_dist_x; i++)
        {
            int xl = __float2int_rn(j * c - i * s + x);
            int zl = __float2int_rn(j * s + i * c + z);

            if (xl < 0 || xl >= width)
                continue;

            if (zl < 0 || zl >= height)
                continue;

            if (xl >= lower_bound_ego_x && xl <= upper_bound_ego_x && zl >= upper_bound_ego_z && zl <= lower_bound_ego_z)
                continue;

            int segmentation_class = __float2int_rn(frame[zl * width + xl].x);

            if (classCost[segmentation_class] < 0)
            {
                // if (x == 115 && z == 16)
                // {
                //     printf("(%d, %d) not feasible on angle %f because of position: (%d, %d)\n", x, z, angle_radians * 180 / HIP_PI_F, xl, zl);
                //     printf("(%d, %d) min distances: W: %d  H: %d\n",  x, z, min_dist_x, min_dist_z);
                // }
                return false;
            }
        }

    // if (x == 115 && z == 16)
    // {
    //     printf("(%d, %d) feasible on angle %f\n", x, z, angle_radians * 180 / HIP_PI_F);
    // }
    return true;
}
