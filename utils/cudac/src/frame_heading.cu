#include "hip/hip_runtime.h"
#include "../include/cuda_basic.h"
#include "../include/class_def.h"
#include <hip/hip_math_constants.h>


__device__ float __CUDA_KERNEL_ComputeHeading(int p1_x, int p1_y, int p2_x, int p2_y, bool *valid, int width, int height)
{
    *valid = false;
    if (p1_x == p2_x && p1_y == p2_y)
        return 0.0;

    if (p1_x < 0 || p1_y < 0 || p2_x < 0 || p2_y < 0)
        return 0.0;

    if (p1_x >= width || p1_y >= height || p2_x >= width || p2_y >= height)
        return 0.0;

    float dx = p2_x - p1_x;
    float dz = p2_y - p1_y;
    *valid = true;
    float heading = HIP_PI_F / 2 - atan2f(-dz, dx);

    if (heading > HIP_PI_F) // greater than 180 deg
        heading = heading - 2 * HIP_PI_F;

    return heading;
}

__device__ float __CUDA_KERNEL_ComputeHeading_Unbound_Values(int p1_x, int p1_y, int p2_x, int p2_y, bool *valid, int width, int height)
{
    *valid = false;
    if (p1_x == p2_x && p1_y == p2_y)
        return 0.0;

    float dx = p2_x - p1_x;
    float dz = p2_y - p1_y;
    *valid = true;
    float heading = HIP_PI_F / 2 - atan2f(-dz, dx);

    if (heading > HIP_PI_F) // greater than 180 deg
        heading = heading - 2 * HIP_PI_F;

    return heading;
}
