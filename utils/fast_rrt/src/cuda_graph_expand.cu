#include "hip/hip_runtime.h"
#include "cuda_basic.h"
#include "class_def.h"
#include <hip/hip_math_constants.h>

__global__ void setup_kernel(hiprandState *state, long long seed){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(1234, idx, 0, &state[idx]);
}