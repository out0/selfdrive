#include "hip/hip_runtime.h"
#include "cuda_basic.h"
#include "class_def.h"
#include <hip/hip_math_constants.h>

#define NUM_POINTS_ON_MEAN 3

__device__ static double __CUDA_KERNEL_ComputeHeading(int p1_x, int p1_y, int p2_x, int p2_y, bool *valid, int width, int height)
{
    *valid = false;
    if (p1_x == p2_x && p1_y == p2_y)
        return 0.0;

    if (p1_x < 0 || p1_y < 0 || p2_x < 0 || p2_y < 0)
        return 0.0;

    if (p1_x >= width || p1_y >= height || p2_x >= width || p2_y >= height)
        return 0.0;

    double dx = p2_x - p1_x;
    double dz = p2_y - p1_y;
    *valid = true;
    double heading = HIP_PIO2_HI - atan2(-dz, dx);

    if (heading > HIP_PI) // greater than 180 deg
        heading = heading - 2 * HIP_PI;

    return heading;
}
__device__ double __CUDA_KERNEL_compute_mean_heading(double4 *waypoints, int pos, int waypoints_count, bool *valid, int width, int height)
{
    double heading = 0.0;
    int count = 0;

    for (int j = 1; j <= NUM_POINTS_ON_MEAN; j++)
    {
        bool v = false;
        if (pos + j >= waypoints_count)
            break;
        heading += __CUDA_KERNEL_ComputeHeading((int)waypoints[pos].x, (int)waypoints[pos].y, (int)waypoints[pos + j].x, (int)waypoints[pos + j].y, &v, width, height);
        if (!v)
            break;
        count++;
    }

    if (count != NUM_POINTS_ON_MEAN)
    {
        count = 0;
        // compute in reverse
        for (int j = 1; j <= NUM_POINTS_ON_MEAN; j++)
        {
            bool v = false;
            if (pos - j < 0)
            {
                *valid = false;
                return 0.0;
            }
            heading += __CUDA_KERNEL_ComputeHeading((int)waypoints[pos - j].x, (int)waypoints[pos - j].y, (int)waypoints[pos].x, (int)waypoints[pos].y, &v, width, height);
            if (!v)
                break;
            count++;
        }
    }

    *valid = count > 0;

    if (*valid)
        return heading / count;

    return 0.0;
}
__device__ static bool __CUDA_KERNEL_ComputeFeasibleForAngle(float3 *frame, int *classCost, double *checkParams, int x, int z, double angle_radians)
{
    int width = __double2int_rn(checkParams[0]);
    int height = __double2int_rn(checkParams[1]);
    int min_dist_x = __double2int_rn(checkParams[2]);
    int min_dist_z = __double2int_rn(checkParams[3]);
    int lower_bound_ego_x = __double2int_rn(checkParams[4]);
    int lower_bound_ego_z = __double2int_rn(checkParams[5]);
    int upper_bound_ego_x = __double2int_rn(checkParams[6]);
    int upper_bound_ego_z = __double2int_rn(checkParams[7]);

    double c = cos(angle_radians);
    double s = sin(angle_radians);

    for (int i = -min_dist_z; i <= min_dist_z; i++)
        for (int j = -min_dist_x; j <= min_dist_x; j++)
        {
            int xl = __double2int_rn(j * c - i * s + x);
            int zl = __double2int_rn(j * s + i * c + z);

            if (xl < 0 || xl >= width)
                continue;

            if (zl < 0 || zl >= height)
                continue;

            if (xl >= lower_bound_ego_x && xl <= upper_bound_ego_x && zl >= upper_bound_ego_z && zl <= lower_bound_ego_z)
                continue;

            int segmentation_class = __double2int_rn(frame[zl * width + xl].x);

            if (classCost[segmentation_class] < 0)
                return false;
        }

    return true;
}
__device__ static double to_radians(double angle)
{
    return (angle * HIP_PI) / 180;
}
__device__ static double to_degrees(double angle)
{
    return (angle * 180) / HIP_PI;
}
__device__ static void convert_to_map_coord(double3 &center, double rate_w, double rate_h, double3 &p)
{
    double x = p.x;
    double z = p.y;

    p.x = (center.x - z) / rate_w;
    p.y = (x - center.y) / rate_h;
}
__device__ static void convert_to_waypoint_coord(double3 &center, double rate_w, double rate_h, double3 &p)
{
    double x = p.x;
    double y = p.y;

    p.x = __double2int_rn(center.y + rate_h * y);
    p.y = __double2int_rn(center.x - rate_w * x);
}

__device__ static double compute_euclidean_dist(double3 &start, double3 &end)
{
    double dx = end.x - start.x;
    double dy = end.y - start.y;
    return sqrt(dx * dx + dy * dy);
}
__device__ static double compute_path_heading(double3 p1, double3 p2)
{
    double dy = p2.y - p1.y;
    double dx = p2.x - p1.x;

    if (dy >= 0 && dx > 0) // Q1
        return atan(dy / dx);
    else if (dy >= 0 && dx < 0) // Q2
        return HIP_PI - atan(dy / abs(dx));
    else if (dy < 0 && dx > 0) // Q3
        return -atan(abs(dy) / dx);
    else if (dy < 0 && dx < 0) // Q4
        return atan(dy / dx) - HIP_PI;
    else if (dx == 0 && dy > 0)
        return HIP_PIO2_HI;
    else if (dx == 0 && dy < 0)
        return -HIP_PIO2_HI;
    return 0.0;
}
__device__ static double clip(double val, double min, double max)
{
    if (val < min)
        return min;
    if (val > max)
        return max;
    return val;
}

__device__ bool check_kinematic_path(float3 *og, int *classCost, double *checkParams, double3 &start, double3 &end)
{
    double distance = compute_euclidean_dist(start, end);


    double3 _center;
    _center.x = checkParams[13];
    _center.y = checkParams[14];
    double _rate_w = checkParams[8];
    double _rate_h = checkParams[9];
    double _max_steering_angle_deg = checkParams[10];
    double _lr = checkParams[11];
    double velocity_meters_per_s = checkParams[12];


    convert_to_map_coord(_center, _rate_w, _rate_h, start);
    convert_to_map_coord(_center, _rate_w, _rate_h, end);
    double dt = 0.1;

    double max_turning_angle = to_radians(_max_steering_angle_deg);
    double heading = to_radians(start.z);

    double path_heading = compute_path_heading(start, end);
    double steering_angle_deg = clip(path_heading - heading, -max_turning_angle, max_turning_angle);
    double ds = velocity_meters_per_s * dt;

    int total_steps = __double2int_rn(round(distance / ds));

    double best_end_dist = distance;
    double x = start.x;
    double y = start.y;

    double3 last_p;
    double3 next_p;

    for (int i = 0; i < total_steps; i++)
    {
        double steer = tan(steering_angle_deg);
        double beta = atan(steer / _lr);

        x += ds * cos(heading + beta);
        y += ds * sin(heading + beta);
        heading += ds * cos(beta) * steer / (2 * _lr);

        next_p.x = x;
        next_p.y = y;
        next_p.z = heading;

        path_heading = compute_path_heading(next_p, end);
        steering_angle_deg = clip(path_heading - heading, -max_turning_angle, max_turning_angle);
        double dist = compute_euclidean_dist(next_p, end);


        convert_to_waypoint_coord(_center, _rate_w, _rate_h, next_p);

        if (next_p.x == last_p.x && next_p.y == last_p.z)
            continue;

        if (best_end_dist < dist)
        {
            return best_end_dist <= 2;
        }

        if (!__CUDA_KERNEL_ComputeFeasibleForAngle(og, classCost, checkParams, next_p.x, next_p.y, next_p.z))
            return false;

        last_p.x = next_p.x;
        last_p.z = next_p.z;

        best_end_dist = dist;
    }
    
    return false;
}

__global__ void CUDA_KERNEL_check_connection_feasible(float3 *og, int *classCost, double *checkParams, unsigned int *pcount, double3 &start, double3 &end) {
    int width = __double2int_rn(checkParams[0]);
    int height = __double2int_rn(checkParams[1]);
    bool res = check_kinematic_path(og, classCost, checkParams, start, end);
    if (res) return;
    atomicInc(pcount, width * height);
}

bool CUDA_check_connection_feasible(float3 *og, int *classCost, double *checkParams, unsigned int *pcount, double3 &start, double3 &end)
{
    int width = static_cast<int>(round(checkParams[0]));
    int height = static_cast<int>(round(checkParams[1]));
    int size = width * height;

    int numBlocks = floor(size / 256) + 1;

    *pcount = 0;
    CUDA_KERNEL_check_connection_feasible<<<numBlocks, 256>>>(og, classCost, checkParams, pcount, start, end);
    CUDA(hipDeviceSynchronize());

    return *pcount == 0;

}


__global__ void CUDA_KERNEL_build_path(double4 * graph, float3 *og, double *checkParams, double3 *inputParams)
{
    int width = checkParams[0];
    int height = checkParams[1];

    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos > width * height)
        return;

    int zt = pos / width;
    int xt = pos - zt * width;

    if (graph[pos].w != 1.0) return;

    double3 start = inputParams[0];
    double3 end = inputParams[1];
    int r,g, b;
    r = __double2int_rd(inputParams[2].x);
    g = __double2int_rd(inputParams[2].y);
    b = __double2int_rd(inputParams[2].z);


    // limits the process to a single thread. I dont want a bunch of threads doing the same thing...
    if (xt != __double2int_rn(start.x) || zt != __double2int_rn(start.y)) 
        return;

    // Now lets build the path
    double distance = compute_euclidean_dist(start, end);   

    double3 _center;
    _center.x = checkParams[13];
    _center.y = checkParams[14];
    double _rate_w = checkParams[8];
    double _rate_h = checkParams[9];
    double _max_steering_angle_deg = checkParams[10];
    double _lr = checkParams[11];
    double velocity_meters_per_s = checkParams[12];

    convert_to_map_coord(_center, _rate_w, _rate_h, start);
    convert_to_map_coord(_center, _rate_w, _rate_h, end);
    double dt = 0.1;

    double max_turning_angle = to_radians(_max_steering_angle_deg);
    double heading = to_radians(start.z);

    double path_heading = compute_path_heading(start, end);
    double steering_angle_deg = clip(path_heading - heading, -max_turning_angle, max_turning_angle);
    double ds = velocity_meters_per_s * dt;

    int total_steps = __double2int_rn(distance / ds);

    double best_end_dist = distance;
    double x = start.x;
    double y = start.y;

    double3 last_p;
    double3 next_p;

    double iL = 1 / (2 * _lr);

    for (int i = 0; i < total_steps; i++)
    {
        double steer = tan(steering_angle_deg);
        double beta = atan(steer / _lr);

        x += ds * cos(heading + beta);
        y += ds * sin(heading + beta);
        heading += ds * cos(beta) * steer * iL;


        next_p.x = x;
        next_p.y = y;
        next_p.z = to_degrees(heading);

        path_heading = compute_path_heading(next_p, end);
        steering_angle_deg = clip(path_heading - heading, -max_turning_angle, max_turning_angle);
        double dist = compute_euclidean_dist(next_p, end);

        convert_to_waypoint_coord(_center, _rate_w, _rate_h, next_p);

        if (next_p.x == last_p.x && next_p.y == last_p.y)
            continue;

        if (best_end_dist < dist)
            return;

        int pos = next_p.y * width + next_p.x;

        og[pos].x = r;
        og[pos].y = g;
        og[pos].z = b;

        last_p.x = next_p.x;
        last_p.y = next_p.y;
        best_end_dist = dist;
    }
}



__global__ void __CUDA_KERNEL_find_nearest_feasible_neighbor_dist(double4 *graph, float3 *og, int *classCost, double *checkParams, int target_x, int target_z, int *bestDistance)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    int width = __double2int_rn(checkParams[0]);
    int height = __double2int_rn(checkParams[1]);

    if (pos > width * height)
        return;

    int z = pos / width;
    int x = pos - z * width;

    if (graph[pos].w != 1.0) // w means that the point is part of the graph
        return;

    int dx = target_x - x;
    int dz = target_z - z;

    // may be optimized with a max distance to check? 
    int dist = __double2int_rn(sqrtf(dx * dx + dz * dz));

    double3 start, end;

    start.x = x;
    start.y = z;
    start.z = graph[pos].z;

    end.x = target_x;
    end.y = target_z;
    end.z = graph[pos].z;


    if (!check_kinematic_path(og, classCost, checkParams, start, end))
        return;

    atomicMin(bestDistance, dist);
}

/*
This method is much less efficient than it's CPU equivalent. It is only used for testing.
*/
void __tst_CUDA_build_path(double4 *graph, float3 *og, double *checkParams, double3 &start, double3 &end, int r, int g, int b)
{
    int width = static_cast<int>(round(checkParams[0]));
    int height = static_cast<int>(round(checkParams[1]));
    int size = width * height;

    int numBlocks = floor(size / 256) + 1;

    double3 *inputParams = nullptr;
    if (!cudaAllocMapped(&inputParams, sizeof(double3) * 3))
        return;

    inputParams[0].x = start.x;
    inputParams[0].y = start.y;
    inputParams[0].z = start.z;
    inputParams[1].x = end.x;
    inputParams[1].y = end.y;
    inputParams[1].z = end.z;
    inputParams[2].x = r;
    inputParams[2].y = g;
    inputParams[2].z = b;
    

    CUDA_KERNEL_build_path<<<numBlocks, 256>>>(graph, og, checkParams, inputParams);
    CUDA(hipDeviceSynchronize());
}