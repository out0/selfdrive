#include "hip/hip_runtime.h"
#include "../include/cuda_basic.h"
#include "../include/class_def.h"
#include <hip/hip_math_constants.h>

__global__ static void __CUDA_KERNEL_Clear(float4 *frame, int width, int height)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos > width * height)
        return;

    frame[pos].x = 0.0;
    frame[pos].y = 0.0;
    frame[pos].z = 0.0;
    frame[pos].w = 0.0;
}

__global__ static void __CUDA_KERNEL_count_elements_in_graph(float4 *frame, int width, int height, unsigned int *count)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos > width * height)
        return;

    if (frame[pos].w == 1.0)
    {
        // int z = pos / width;
        // int x = pos - z * width;
        // printf("%d, %d is in graph, inc count...\n", x, z);
        atomicInc(count, width * height);
    }
}

__global__ static void __CUDA_KERNEL_check_in_graph(float4 *frame, int width, int height, int target_x, int target_z, unsigned int *count)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos > width * height)
        return;

    int z = pos / width;
    int x = pos - z * width;

    if (x != target_x || z != target_z)
        return;

    if (frame[pos].w == 1.0)
    {
        // printf("check in graph success for %d, %d\n", x, z);
        atomicInc(count, width * height);
    }
}

void CUDA_clear(float4 *frame, int width, int height)
{
    int size = width * height;

    int numBlocks = floor(size / 256) + 1;

    __CUDA_KERNEL_Clear<<<numBlocks, 256>>>(frame, width, height);

    CUDA(hipDeviceSynchronize());
}

/*
    BEST NEIGHBOR
*/

__global__ void __CUDA_KERNEL_find_best_neighbor_cost(float4 *frame, int width, int height, int target_x, int target_z, float radius, int *bestCost)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos > width * height)
        return;

    int z = pos / width;
    int x = pos - z * width;

    if (frame[pos].w != 1.0) // w means that the point is part of the graph
        return;

    int dx = target_x - x;
    int dz = target_z - z;

    int dist = dx * dx + dz * dz;

    int r = radius * radius;

    if (dist > r)
        return;

    // self cost + dist
    int cost =__float2int_rn(sqrtf(dist) + frame[pos].z);

    atomicMin(bestCost, cost);
}

__global__ void __CUDA_KERNEL_find_waypoint_with_best_cost(float4 *frame, int width, int height, int target_x, int target_z, float radius, int *bestCost, int3 *point)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos > width * height)
        return;

    int z = pos / width;
    int x = pos - z * width;

    if (frame[pos].w != 1.0) // z means that the point is part of the graph
        return;

    int dx = target_x - x;
    int dz = target_z - z;

    int dist = dx * dx + dz * dz;
    int cost =__float2int_rn(sqrtf(dist) + frame[pos].z);


    if (cost == *bestCost)
    {
        point->x = x;
        point->y = z;
        point->z = 1;
    }
}

int *CUDA_find_best_neighbor(float4 *frame, int3 *point, int width, int height, int goal_x, int goal_z, float radius)
{
    int size = width * height;

    int numBlocks = floor(size / 256) + 1;

    int *bestCost;
    if (!cudaAllocMapped(&bestCost, sizeof(int)))
    {
        fprintf(stderr, "[CUDA RRT] unable to allocate %ld bytes for best cost in CUDA_find_best_neighbor()\n", sizeof(int));
        return nullptr;
    }

    *bestCost = 999999999;
    __CUDA_KERNEL_find_best_neighbor_cost<<<numBlocks, 256>>>(frame, width, height, goal_x, goal_z, radius, bestCost);
    CUDA(hipDeviceSynchronize());

    if (*bestCost < 999999999)
    {
        __CUDA_KERNEL_find_waypoint_with_best_cost<<<numBlocks, 256>>>(frame, width, height, goal_x, goal_z, radius, bestCost, point);
        CUDA(hipDeviceSynchronize());
    }

    int *res = nullptr;

    if (point->z >= 1.0)
        res = new int[3]{point->x, point->y, 1};
    else
        res = new int[3]{0, 0, 0};

    hipHostFree(bestCost);

    return res;
}

/*
    NEAREST NEIGHBOR
*/

__global__ void __CUDA_KERNEL_find_nearest_neighbor_dist(float4 *frame, int width, int height, int target_x, int target_z, int *bestDistance)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos > width * height)
        return;

    int z = pos / width;
    int x = pos - z * width;

    if (frame[pos].w != 1.0) // w means that the point is part of the graph
        return;

    int dx = target_x - x;
    int dz = target_z - z;

    int dist = dx * dx + dz * dz;

    atomicMin(bestDistance, dist);
}

__global__ void __CUDA_KERNEL_find_waypoint_with_nearest_dist(float4 *frame, int width, int height, int target_x, int target_z, int *bestDistance, int3 *point)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos > width * height)
        return;

    int z = pos / width;
    int x = pos - z * width;

    if (frame[pos].w != 1.0) // z means that the point is part of the graph
        return;

    int dx = target_x - x;
    int dz = target_z - z;

    int dist = dx * dx + dz * dz;

    if (dist == *bestDistance)
    {
        point->x = x;
        point->y = z;
        point->z = 1;
    }
}

int *CUDA_find_nearest_neighbor(float4 *frame, int3 *point, int width, int height, int goal_x, int goal_z)
{
    int size = width * height;

    int numBlocks = floor(size / 256) + 1;

    int *bestCost;
    if (!cudaAllocMapped(&bestCost, sizeof(int)))
    {
        fprintf(stderr, "[CUDA RRT] unable to allocate %ld bytes for best cost in CUDA_find_best_neighbor()\n", sizeof(int));
        return nullptr;
    }

    *bestCost = 999999999;
    __CUDA_KERNEL_find_nearest_neighbor_dist<<<numBlocks, 256>>>(frame, width, height, goal_x, goal_z, bestCost);
    CUDA(hipDeviceSynchronize());

    if (*bestCost < 999999999)
    {
        __CUDA_KERNEL_find_waypoint_with_nearest_dist<<<numBlocks, 256>>>(frame, width, height, goal_x, goal_z, bestCost, point);
        CUDA(hipDeviceSynchronize());
    }

    int *res = nullptr;

    if (point->z >= 1.0)
        res = new int[3]{point->x, point->y, 1};
    else
        res = new int[3]{0, 0, 0};

    hipHostFree(bestCost);

    return res;
}

unsigned int CUDA_count_elements_in_graph(float4 *frame, int width, int height)
{
    int size = width * height;

    int numBlocks = floor(size / 256) + 1;

    unsigned int *count;
    if (!cudaAllocMapped(&count, sizeof(unsigned int)))
    {
        fprintf(stderr, "[CUDA RRT] unable to allocate %ld bytes for counting elements in graph\n", sizeof(unsigned int));
        return 0;
    }

    *count = 0;
    __CUDA_KERNEL_count_elements_in_graph<<<numBlocks, 256>>>(frame, width, height, count);
    CUDA(hipDeviceSynchronize());

    unsigned int res = *count;

    // printf ("CUDA_count_elements_in_graph => %d\n", res);

    hipHostFree(count);
    return res;
}

bool CUDA_check_in_graph(float4 *frame, int width, int height, int x, int z)
{
    int size = width * height;

    int numBlocks = floor(size / 256) + 1;

    unsigned int *count;
    if (!cudaAllocMapped(&count, sizeof(unsigned int)))
    {
        fprintf(stderr, "[CUDA RRT] unable to allocate %ld bytes for counting elements in graph\n", sizeof(unsigned int));
        return 0;
    }

    *count = 0;
    __CUDA_KERNEL_check_in_graph<<<numBlocks, 256>>>(frame, width, height, x, z, count);
    CUDA(hipDeviceSynchronize());

    int res = *count;

    hipHostFree(count);
    return res > 0;
}


__global__ static void __CUDA_KERNEL_list_elements_in_graph(float4 *frame, float *res, int width, int height, unsigned int *list_pos)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos > width * height)
        return;

    int z = pos / width;
    int x = pos - z * width;

    if (frame[pos].w == 1.0)
    {
        int store_pos = 5 * atomicInc(list_pos, width * height);
        res[store_pos] = x;
        res[store_pos+1] = z;
        res[store_pos+2] = frame[pos].x;
        res[store_pos+3] = frame[pos].y;
        res[store_pos+4] = frame[pos].z;
    }
}


void CUDA_list_elements(float4 *frame, 
    float *result,
    int width,
    int height,
    int count)
{

    int size = width * height;

    int numBlocks = floor(size / 256) + 1;

    float *cudaResult;
    if (!cudaAllocMapped(&cudaResult, sizeof(float) * count * 5))
    {
        fprintf(stderr, "[CUDA Graph] unable to allocate %ld bytes for list elements in CUDA_list_elements()\n", sizeof(float) * count * 5);
        return;
    }

    unsigned int *listPos;
    if (!cudaAllocMapped(&listPos, sizeof(unsigned int)))
    {
        fprintf(stderr, "[CUDA Graph] unable to allocate %ld bytes for list elements position in CUDA_list_elements()\n", sizeof(unsigned int));
        hipHostFree(cudaResult);
        return;
    }
    *listPos = 0;

    __CUDA_KERNEL_list_elements_in_graph<<<numBlocks, 256>>>(
        frame,
        cudaResult, 
        width, 
        height, 
        listPos);

    CUDA(hipDeviceSynchronize());

    for (int i = 0; i < *listPos * 5; i++) {
        result[i] = cudaResult[i];
    }

    hipHostFree(cudaResult);
    hipHostFree(listPos);

}